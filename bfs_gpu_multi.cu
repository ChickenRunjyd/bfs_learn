/**
 File name: bfs_gpu_multi.cu
 Author: Yuede Ji
 Last update: 9:54 10-03-2015
 Description: Using multi thread to implent GPU version of bfs.
    Calculate the shortest distance between each other

**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024 //1024 vertex number
//Using arrays to implement queue
#define N_block 32
#define imax(a, b) (a>b?a:b)
char filein[] = "/home/yuede/dataset/kron_10_4.dat";
char fileout[] = "/home/yuede/dataset/kron_10_4.m_gpu";
//Using arrays to implement queue
//int q[N];

int edge[N][N];
//int visit[N];
int dist[N][N];

__global__ void bfs(int *edg, int *dis)
{
    int q[N];
    int vis[N];
    memset(vis, 0, N*sizeof(int));
    memset(q, 0, N*sizeof(int));
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    q[0] = index;
    vis[index] = 1;
    int l = 1; // record the size of the queue
    int front = 0; // identify the front element
    int end = 0; // identify the end element
    printf("index = %d\n", index);
    while(l>0)
    {
        int cur = q[front];
        ++front;
        --l;
        if(front >= N)
            front -= N;
        
        for(int i=0; edg[cur*N + i]!=0; ++i)
        {
            int v = edg[cur*N + i];
            printf("vis[%d] = %d\n", v, vis[v]);
            if(vis[v])
                continue;
            //printf("edg[cur*N + i] = %d\n", edg[cur*N + i]);
            dis[index*N + v] = dis[index*N + cur] + 1;
            //printf("dis[%d] = %d\n", v, dis[v]);
            ++end;
            if(end >= N)
                end -= N;
            q[end] = v;
            vis[v] = 1;
            ++l;
        }
    }
    printf("index = %d finished\n", index);
}
int main()
{
    FILE *fp_in = fopen(filein, "r");
    int v, e;
    int num_v=0;
    memset(edge, 0, N*N*sizeof(int));
    while(fscanf(fp_in, "%d %d", &v, &e)!=EOF)
    {
        ++num_v;
        for(int i=0; i<e; ++i)
        {
            int v1;
            fscanf(fp_in, "%d", &v1);
            edge[v][i] = v1;//v->v1
        }
    }
    fclose(fp_in);

    int *dev_edge;
    int *dev_dist;
        
    //allocate memory on GPU
    hipMalloc( (void **) &dev_edge, N*N*sizeof(int));
    hipMalloc( (void **) &dev_dist, N*N*sizeof(int));

    //initialize GPU memory
    hipMemset( dev_dist, 0, N*N*sizeof(int));
    
    //copy edge from CPU to GPU
    hipMemcpy(dev_edge, edge, N*N*sizeof(int), hipMemcpyHostToDevice);
    
    bfs<<<N_block, (N+N_block-1)/N_block>>>(dev_edge, dev_dist);
    //bfs<<<1, 1>>>(dev_edge, dev_dist);
    hipMemcpy(dist, dev_dist, N*N*sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_edge);
    hipFree(dev_dist);

    FILE *fp_out = fopen(fileout, "w");
    for(int i=0; i<num_v; ++i)
    {
        fprintf(fp_out, "%d", i);
        for(int j=0; j<num_v; ++j)
            fprintf(fp_out, " %d", imax(dist[i][j], dist[j][i]));
        fprintf(fp_out, "\n");
    }
    fclose(fp_out);
    printf("Finished!\n");
    return 0;
}
