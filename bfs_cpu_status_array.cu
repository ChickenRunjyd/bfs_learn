/**
 File name: bfs_cpu_status_array.cu
 Author: Yuede Ji
 Last update: 11:00 10-09-2015
 Description: Using status array to implent CPU version of bfs.
    Calculate the shortest distance from 0 to others
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//Using arrays to implement queue
char filein[] = "/home/yuede/dataset/kron_16_16.dat";// no need
char fileout[] = "/home/yuede/dataset/kron_16_16.cpu.as.result";
char file_v_e[] = "/home/yuede/dataset/kron_16_16.v_e";
char file_beg_pos[] = "/home/yuede/dataset/kron_16_16.beg.pos";
char file_csr[] = "/home/yuede/dataset/kron_16_16.csr";

/**int *beg_pos;
int *csr;
int *sa;
**/
const int v_num = 65535;
const int e_num = 2097152;
const int INF = 0x7FFFFFFF; 
int beg_pos[v_num+1];
int csr[e_num];
int sa[v_num];
//load from .dat files, and store in array csr[N*N], beg_pos[N]
int csr_begin(int v, int e)
{
    /**
    int v, e;
    FILE * fp_v_e = fopen(file_v_e, "r");
    fscanf(fp_v_e, "%d%d", &v, &e);
    fclose(fp_v_e);

    beg_pos = (int *) malloc(v+1);
    csr = (int *) malloc(e);
    **/   
    FILE * fp_beg = fopen(file_beg_pos, "r");
    int i = 0;
    int p;
    while(fscanf(fp_beg, "%d", &p) != EOF)
    {
        beg_pos[i] = p;
        ++i;
    }
    fclose(fp_beg);

    i = 0;
    FILE * fp_csr = fopen(file_csr, "r");
    while(fscanf(fp_csr, "%d", &p) != EOF)
    {
        csr[i] = p;
        ++i;
    }
    fclose(fp_csr);
    printf("i=%d\n", i);
    return v;
}
void bfs_sa(int root, int v)
{
    for(int i=0; i<v; ++i)
        sa[i] = INF;
    int count = 1;
    int level = 0;
    sa[0] = 0;
    bool flag; //flag whether current level has nodes
    while(count < v)
    {
        flag = false;
        for(int i=0; i<v; ++i)
        {
            if(sa[i] == level)///node i belongs to current level
            {
                if(!flag)
                    flag = true;
                for(int j=beg_pos[i]; j<beg_pos[i+1]; ++j)
                {
                    if(sa[csr[j]] <= level + 1)
                        continue;
                    sa[csr[j]] = level + 1;
                    ++count;
                    //printf("count = %d\n", count);
                }                    
            }
        }
        ++level;
        //printf("level = %d\n", level);
        if(!flag)//indicates current level has no vertex
            break;
    }
}
int main()
{
    csr_begin(v_num, e_num);

    bfs_sa(0, v_num);

    FILE * fp_out = fopen(fileout, "w");

    for(int i=0; i<v_num; ++i)
        fprintf(fp_out, "%d\n", sa[i]);
    fclose(fp_out);
    
    return 0;
}

