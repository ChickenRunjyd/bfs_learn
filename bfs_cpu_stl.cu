/**
 File name: bfs_cpu_stl.cu
 Author: Yuede Ji
 Last update: 10:27 10-02-2015
 Description: Using stl queue to implement the easiest version of bfs.

**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <queue>
#include <stdlib.h>
#include <string.h>

using namespace std;

#define N 1025

char filein[] = "/home/yuede/dataset/kron_10_4.dat";
char fileout[] = "/home/yuede/dataset/kron_10_4.stl";

queue<int> q;

int edge[N][N];
int visit[N];
int dist[N];
int bfs(int root)
{
    memset(dist, 0, sizeof(int) * N);
    q.push(root);
    while(!q.empty())
    {
        int bottom = q.front();
        q.pop();
        for(int i=0; edge[bottom][i]!=0; ++i)
        {
            int v = edge[bottom][i];
            if(visit[v])
                continue;
            dist[v] = dist[bottom] + 1;
            q.push(v);
            visit[v] = 1;
        }
    }
    return 0;
}
int main()
{
    FILE *fp_in = fopen(filein, "r");
    ///fscanf(fp, "%d", &n);
    ///printf("%d\n", n); 
    int v, e;
    int num_v=0;
    memset(edge, 0, N*N*sizeof(int));
    memset(visit, 0, N*sizeof(int));
    while(fscanf(fp_in, "%d %d", &v, &e)!=EOF)
    {
        ++num_v;
        for(int i=0; i<e; ++i)
        {
            int v1;
            fscanf(fp_in, "%d", &v1);
            edge[v][i] = v1;//v->v1
        }
    }
    fclose(fp_in);
    bfs(0);
    //printf("num_v = %d\n", num_v);
    FILE * fp_out = fopen(fileout, "w");
    for(int i=0; i<num_v; ++i)
        fprintf(fp_out, "distance[0][%d] = %d\n", i, dist[i]);
    fclose(fp_out);
    printf("Finished!\n");
    return 0;
}
