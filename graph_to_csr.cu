/**
 File name: graph_to_csr.cu
 Author: Yuede Ji
 Last update: 15:52 10-09-2015
 Description: convert current normal graph file to scr and begin position stored file
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 65536 // vertex number
//Using arrays to implement queue
/**
char filein[] = "/home/yuede/dataset/kron_16_16.dat";
char fileout[] = "/home/yuede/dataset/kron_16_16.cpu.as.result";
char file_beg_pos[] = "/home/yuede/dataset/kron_16_16.beg.pos";
char file_csr[] = "/home/yuede/dataset/kron_16_16.csr";
char file_v_e[] = "/home/yuede/dataset/kron_16_16.v_e";
**/
char filein[] = "/home/yuede/dataset/kron_10_4.dat";
char fileout[] = "/home/yuede/dataset/kron_10_4.cpu.as.result";
char file_beg_pos[] = "/home/yuede/dataset/kron_10_4.beg.pos";
char file_csr[] = "/home/yuede/dataset/kron_10_4.csr";
char file_v_e[] = "/home/yuede/dataset/kron_10_4.v_e";

const int INF = 0x7FFFFFFF; 

int v_num = 0;
int e_num = 0;
void empty_file(char * filename)
{
    FILE * fp = fopen(filename, "w");
    fclose(fp);
}
//load from .dat files, and store in array csr[N*N], beg_pos[N]
void csr_begin(char *filename)
{
    empty_file(file_beg_pos);
    empty_file(file_csr);
    FILE * fp_in = fopen(filein, "r");
    FILE * fp_csr = fopen(file_csr, "a");
    FILE * fp_beg_pos = fopen(file_beg_pos, "a");
    int v, n;//v denotes current vertex, n denotes no. of adjacent node
    int j = 0;// j denotes the index in csr[N*N];
    int begin = 0;

    fprintf(fp_beg_pos, "%d\n", begin);
    while(fscanf(fp_in, "%d%d", &v, &n)!=EOF)
    {
        //printf("%d %d\n", v, n);
        begin += n;
        fprintf(fp_beg_pos, "%d\n", begin);
        for(int i=0; i<n; ++i)
        {
            fscanf(fp_in, "%d", &j);
            fprintf(fp_csr, "%d\n", j);
        }
    }
    fclose(fp_beg_pos);
    fclose(fp_csr);
    e_num = begin+1;
    v_num = v+1;
    printf("v_num = %d, e_num = %d\n", v_num, e_num);
    FILE *fp_v_e = fopen(file_v_e, "w");
    fprintf(fp_v_e, "%d %d\n", v_num, e_num);
    fclose(fp_v_e);
}
int main()
{
    csr_begin(filein);
    return 0;
}

