/**
 File name: bfs_gpu_status_array_10_4.cu
 Author: Yuede Ji
 Last update: 21:22 10-11-2015
 Description: Using status array to implent GPU version of bfs.
    Calculate the shortest distance from 0 to others
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//Using arrays to implement queue
/**char filein[] = "/home/yuede/dataset/kron_16_16.dat";// no need
char fileout[] = "/home/yuede/dataset/kron_16_16.gpu.as.result";
char file_v_e[] = "/home/yuede/dataset/kron_16_16.v_e";
char file_beg_pos[] = "/home/yuede/dataset/kron_16_16.beg.pos";
char file_csr[] = "/home/yuede/dataset/kron_16_16.csr";
**/
char filein[] = "/home/yuede/dataset/kron_10_4.dat";// no need
char fileout[] = "/home/yuede/dataset/kron_10_4.gpu.as.result";
char file_v_e[] = "/home/yuede/dataset/kron_10_4.v_e";
char file_beg_pos[] = "/home/yuede/dataset/kron_10_4.beg.pos";
char file_csr[] = "/home/yuede/dataset/kron_10_4.csr";


const int v_num = 1024;
const int e_num = 8193;
const int INF = 0x7FFFFFFF;
const int threads_num = 32;

int beg_pos[v_num+1];
int csr[e_num];
int sa[v_num];
//load from .dat files, and store in array csr[N*N], beg_pos[N]

int csr_begin(int v, int e);
void bfs_sa(int root, int v, int e);
__global__ void traverse_one(int level, int * dev_sa, int * dev_beg_pos, int * dev_csr, int dev_flag)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(dev_sa[id] == level)///node i belongs to current level
    {
        //int j = dev_beg_pos[id];
        for(int j=dev_beg_pos[id]; j<dev_beg_pos[id+1]; ++j)
        {
            if(dev_sa[dev_csr[j]] > level + 1)
            {  
                printf("dev_csr[%d] = %d\n", j, dev_csr[j]);
                dev_sa[dev_csr[j]] = level + 1;
                if(!dev_flag)
                    dev_flag = true;
            }
        }                    
    }
}

int main()
{
    csr_begin(v_num, e_num);

    bfs_sa(0, v_num, e_num);

    FILE * fp_out = fopen(fileout, "w");

    for(int i=0; i<v_num; ++i)
        fprintf(fp_out, "%d\n", sa[i]);
    fclose(fp_out);
    
    return 0;
}
void bfs_sa(int root, int v, int e)
{
    for(int i=0; i<v; ++i)
        sa[i] = INF;
    int level = 0;
    sa[0] = 0;
    bool flag = true; //flag whether current level has nodes
    
    int *dev_sa;
    int *dev_beg_pos;
    int *dev_csr;

    for(int i=0; i<10; ++i)
        printf("csr[%d] = %d\n", i, csr[i]);

    hipMalloc( (void **) &dev_sa, v*sizeof(int));
    hipMalloc( (void **) &dev_beg_pos, (v+1)*sizeof(int));
    hipMalloc( (void **) &dev_csr, e*sizeof(int));

    hipMemcpy(dev_sa, sa, v*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_beg_pos, beg_pos, (v+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_csr, csr, e*sizeof(int), hipMemcpyHostToDevice);
    
    bool dev_flag;
    hipMalloc( (void **) &dev_flag, sizeof(bool));

    while(flag)
    {
        flag = false;
        hipMemcpy(&dev_flag, &flag, sizeof(bool), hipMemcpyHostToDevice);
        traverse_one<<<threads_num, threads_num>>>(level, dev_sa, dev_beg_pos, dev_csr, dev_flag);
        hipMemcpy(&flag, &dev_flag, sizeof(bool), hipMemcpyDeviceToHost);
        ++level;
    }
    hipMemcpy(sa, dev_sa, v*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_sa);
    hipFree(dev_beg_pos);
    hipFree(dev_csr);

}

int csr_begin(int v, int e)
{
    FILE * fp_beg = fopen(file_beg_pos, "r");
    int i = 0;
    int p;
    while(fscanf(fp_beg, "%d", &p) != EOF)
    {
        beg_pos[i] = p;
        ++i;
    }
    fclose(fp_beg);

    i = 0;
    FILE * fp_csr = fopen(file_csr, "r");
    while(fscanf(fp_csr, "%d", &p) != EOF)
    {
        csr[i] = p;
        ++i;
    }
    fclose(fp_csr);
    return v;
}
