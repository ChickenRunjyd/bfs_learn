/**
 File name: bfs_cpu_array_multi.cu
 Author: Yuede Ji
 Last update: 21:54 10-02-2015
 Description: Using array to implent CPU version of bfs.
    Calculate the shortest distance between each other 
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <queue>
#include <stdlib.h>
#include <string.h>

using namespace std;
#define N 1024 // vertex number
//Using arrays to implement queue
#define imax(a, b) (a>b?a:b)

char filein[] = "/home/yuede/dataset/edge.dat";
char fileout[] = "/home/yuede/dataset/edge.cpu_multi";
//Using arrays to implement queue
int q[N];

int edge[N][N];
int visit[N];
int dist[N][N];

int bfs(int root)
{
    //memset(dist, 0, sizeof(int) * N * N);
    memset(visit, 0, sizeof(int) * N);
    q[0] = root;
    int l = 1; // record the size of the queue
    int front = 0; // identify the front element
    int end = 0; // identify the end element
    while(l>0)
    {
        int cur = q[front];
        ++front;
        --l;
        if(front >= N)
            front %= N;
        
        for(int i=0; edge[cur][i]!=0; ++i)
        {
            int v = edge[cur][i];
            if(visit[v])
                continue;
            dist[root][v] = dist[root][cur] + 1;
            ++end;
            if(end >= N)
                end %= N;
            q[end] = v;
            visit[v] = 1;
            ++l;
        }
    }
    return 0;
}
int main()
{
    FILE *fp_in = fopen(filein, "r");
    int v, e;
    int num_v=0;
    memset(edge, 0, N*N*sizeof(int));
    memset(visit, 0, N*sizeof(int));
    memset(dist, 0, N*N*sizeof(int));
    while(fscanf(fp_in, "%d %d", &v, &e)!=EOF)
    {
        ++num_v;
        for(int i=0; i<e; ++i)
        {
            int v1;
            fscanf(fp_in, "%d", &v1);
            edge[v][i] = v1;//v->v1
        }
    }
    fclose(fp_in);

    for(int i=0; i<num_v; ++i)
        bfs(i);
    FILE *fp_out = fopen(fileout, "w");
    //fprintf("num_v = %d\n", num_v);
    for(int i=0; i<num_v; ++i)
    {
        fprintf(fp_out, "%d", i);
        for(int j=0; j<num_v; ++j)
            fprintf(fp_out, " %d", imax(dist[i][j], dist[j][i]));
        fprintf(fp_out, "\n");
    }
    fclose(fp_out);
    printf("Finished!\n");
    return 0;
}
