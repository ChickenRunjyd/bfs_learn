/**
 File name: bfs_cpu_stl.cu
 Author: Yuede Ji
 Last update: 18:25 10-02-2015
 Description: Using array to implent CPU version of bfs.
    Calculate the shortest distance from 0 to others
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <queue>
#include <stdlib.h>
#include <string.h>

using namespace std;
#define N 1025 // vertex number
//Using arrays to implement queue


char filein[] = "/home/yuede/dataset/kron_10_4.dat";
char fileout[] = "/home/yuede/dataset/kron_10_4.result";
//Using arrays to implement queue
int q[N];

int edge[N][N];
int visit[N];
int dist[N];

int bfs(int root)
{
    memset(dist, 0, sizeof(int) * N);
    q[0] = root;
    int l = 1; // record the size of the queue
    int front = 0; // identify the front element
    int end = 0; // identify the end element
    while(l>0)
    {
        int cur = q[front];
        ++front;
        --l;
        if(front >= N)
            front %= N;
        
        for(int i=0; edge[cur][i]!=0; ++i)
        {
            int v = edge[cur][i];
            if(visit[v])
                continue;
            dist[v] = dist[cur] + 1;
            ++end;
            if(end >= N)
                end %= N;
            q[end] = v;
            visit[v] = 1;
            ++l;
        }
    }
    return 0;
}
int main()
{
    FILE *fp_in = fopen(filein, "r");
    int v, e;
    int num_v=0;
    memset(edge, 0, N*N*sizeof(int));
    memset(visit, 0, N*sizeof(int));
    while(fscanf(fp_in, "%d %d", &v, &e)!=EOF)
    {
        ++num_v;
        for(int i=0; i<e; ++i)
        {
            int v1;
            fscanf(fp_in, "%d", &v1);
            edge[v][i] = v1;//v->v1
        }
    }
    fclose(fp_in);

    bfs(0);
    FILE *fp_out = fopen(fileout, "w");
    //fprintf("num_v = %d\n", num_v);
    for(int i=0; i<num_v; ++i)
        fprintf(fp_out, "distance[0][%d] = %d\n", i, dist[i]);
    fclose(fp_out);
    printf("Finished!\n");
    return 0;
}
