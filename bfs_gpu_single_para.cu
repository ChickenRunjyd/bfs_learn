#include "hip/hip_runtime.h"
/**
 File name: bfs_gpu_single_para.cu
 Author: Yuede Ji
 Last update: 13:38 10-08-2015
 Description: Using single thread to implent parallel GPU version of bfs.
    Calculate the shortest distance from 0 to others
**/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

using namespace std;
#define N 1024 // vertex number
//Using arrays to implement queue


char filein[] = "/home/yuede/dataset/edge.dat";
char fileout[] = "/home/yuede/dataset/edge.s_gpu";
//Using arrays to implement queue
//int q[N];

int edge[N][N];
int visit[N];
int dist[N];

/** Deal the current node, and put his children nodes in the queue.
**/
__device__ void deal_one_node(int cur, int length, int *edg, int *q)
{
    int index = cur + threadIdx.x + blockIdx.x * blockDim.x;
    if(index >= N)
        index -= N;
    for(int i=0; edg[index*N + i]!=0; ++i)
    {
        int v = edg[cur*N + i];
        if(vis[v])
            continue;
        ++l;
        ++end;
        if(end >= N)
            end -= N;
        q[end] = v;
        vis[v] = 1;
    } 
}

__global__ void bfs(int *q, int *edg, int *vis, int *dis)
{
    q[0] = 0;
    int l = 1; // record the size of the queue
    int front = 0; // identify the front element
    int end = 0; // identify the end element
    while(l>0)
    {
        int cur = q[front];
        ++front;
        --l;
        if(front >= N)
            front -= N;

        deal_one_node<<<1, 1>>>(cur, l, edg, q);
        /**
        for(int i=0; edg[cur*N + i]!=0; ++i)
        {
            int v = edg[cur*N + i];
            if(vis[v])
                continue;
            //printf("edg[cur*N + i] = %d\n", edg[cur*N + i]);
            dis[v] = dis[cur] + 1;
            printf("dis[%d] = %d\n", v, dis[v]);
            ++end;
            if(end >= N)
                end -= N;
            q[end] = v;
            vis[v] = 1;
            ++l;
        }
        **/
    }
}
int main()
{
    FILE *fp_in = fopen(filein, "r");
    int v, e;
    int num_v=0;
    memset(edge, 0, N*N*sizeof(int));
    memset(visit, 0, N*sizeof(int));
    while(fscanf(fp_in, "%d %d", &v, &e)!=EOF)
    {
        ++num_v;
        for(int i=0; i<e; ++i)
        {
            int v1;
            fscanf(fp_in, "%d", &v1);
            edge[v][i] = v1;//v->v1
        }
    }
    fclose(fp_in);

    int *dev_edge;
    int *dev_visit;
    int *dev_dist;
    int *q;
        
    //allocate memory on GPU
    hipMalloc( (void **) &dev_edge, N*N*sizeof(int));
    hipMalloc( (void **) &dev_visit, N*N*sizeof(int));
    hipMalloc( (void **) &dev_dist, N*sizeof(int));
    hipMalloc( (void **) &q, N*sizeof(int));

    //initialize GPU memory
    hipMemset( dev_visit, 0, N*N*sizeof(int));
    hipMemset( dev_dist, 0, N*sizeof(int));
    
    //copy edge from CPU to GPU
    hipMemcpy(dev_edge, edge, N*N*sizeof(int), hipMemcpyHostToDevice);
    
    bfs<<<1, 1>>>(q, dev_edge, dev_visit, dev_dist);
    hipMemcpy(dist, dev_dist, N*sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_edge);
    hipFree(dev_visit);
    hipFree(dev_dist);

    FILE *fp_out = fopen(fileout, "w");
    for(int i=0; i<num_v; ++i)
        fprintf(fp_out, "distance[0][%d] = %d\n", i, dist[i]);
    fclose(fp_out);
    printf("Finished!\n");
    return 0;
}
